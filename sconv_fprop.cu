#include "sconv.h"

bool fprop(const float *I, const float *F, float *O,
  unsigned int N, unsigned int C, unsigned int K,
  unsigned int D, unsigned int H, unsigned int W,
  unsigned int R, unsigned int S, unsigned int T,
  unsigned int M, unsigned int P, unsigned int Q,
  unsigned int str_d, unsigned int str_h, unsigned int str_w,
  unsigned int pad_d, unsigned int pad_h, unsigned int pad_w) {
  float alpha = 1.0f;
  unsigned int WN, HWN, DHWN, KRST, RST, RS;
  unsigned int PQ, QN, PQN, MPQN;
  unsigned int magic_RS, magic_S;
  unsigned int shift_RS, shift_S;
  unsigned int magic_Q, shift_Q, magic_PQ, shift_PQ;

  WN = W * N;
  HWN = H * WN;
  DHWN = D * HWN;
  RS = R * S;
  RST = T * RS;
  KRST = K * RST;

  QN = Q * N;
  PQN = P * QN;
  MPQN = M * PQN;
  PQ = P * Q;

  magic64(Q, magic_Q, shift_Q);
  magic64(PQ, magic_PQ, shift_PQ);
  magic32(RST + 32, RS, magic_RS, shift_RS);
  magic32(RS + 32, S, magic_S, shift_S);

  float *test_param;
  hipError_t result;
  result = hipMalloc((void**)&test_param, sizeof(float) * 1024);
  hipMemset(test_param, 0, sizeof(float) * 1024);
  if (result != hipSuccess) {
    std::cerr << "cuda malloc error!" << std::endl;
    exit(1);
  }
  void *args[37] = {&test_param, &O, &I, &F, &alpha,
    &N, &K, &D, &H, &W, &WN, &HWN, &DHWN,
    &C, &KRST, &RST, &RS, &magic_RS, &shift_RS, &S, &magic_S, &shift_S,
    &pad_d, &pad_h, &pad_w, &str_d, &str_h, &str_w,
    &Q, &PQ, &QN, &PQN, &MPQN, &magic_Q, &shift_Q, &magic_PQ, &shift_PQ};
  int gridMPQ = M * P * Q;
  int gridX = gridMPQ;
  int gridY = K / 64 + (K % 64 != 0);
  int gridZ = N / 64 + (N % 64 != 0);

  std::string name = "sconv_fprop_K64_N64";
  hipError_t res = hipModuleLaunchKernel(nervana_kernels[name], gridX, gridY, gridZ, 64, 1, 1, R * S * T * 4 * 2, 0, args, NULL);
  if (res != hipSuccess) {
    std::cerr << "Error launching kernel " << name << " " << res << std::endl;
    return false;
  }
  hipCtxSynchronize();
  //float* h_test = (float *)malloc(sizeof(float) * 64);
  //for (int i = 0; i < 64; ++i) {
  //  std::cout << h_test[i] << " ";
  //}
  //std::cout << std::endl;
  //result = hipMemcpy(h_test, test_param, sizeof(float) * 64, hipMemcpyDeviceToHost);
  //if (result != hipSuccess) {
  //  std::cout << result << std::endl;
  //  std::cerr << "memcpy error!" << std::endl;
  //}

  //for (int i = 0; i < 64; ++i) {
  //  std::cout << h_test[i] << " ";
  //}

  //free(h_test);

  //std::cout << std::endl;

  return true;
}

int main() {
  hipFree(0);
  float *d_I, *d_F, *d_O;
  unsigned int N = 64, C = 1, K = 64, D = 1, H = 5, W = 5, T = 1, R = 5, S = 5;
  unsigned int str_d = 1, str_h = 1, str_w = 1;
  unsigned int pad_d = 0, pad_h = 0, pad_w = 0;
  unsigned int M, P, Q;
  M = (D - T + 2 * pad_d) / str_d + 1;
  P = (H - R + 2 * pad_h) / str_h + 1;
  Q = (W - S + 2 * pad_w) / str_w + 1;
  std::cout << " M " << M << std::endl;
  std::cout << " P " << P << std::endl;
  std::cout << " Q " << Q << std::endl;

  float *h_I = (float *)malloc(N * C * D * H * W * sizeof(float));
  for (int i = 0; i < N * C * D * H * W; ++i) {
    h_I[i] = 1;
  }
  float *h_F = (float *)malloc(K * R * S * T * sizeof(float));
  for (int i = 0; i < K * R * S * T; ++i) {
    h_F[i] = 1;
  }

  hipMalloc((void**)&d_I, sizeof(float) * N * C * D * H * W);
  hipMalloc((void**)&d_F, sizeof(float) * K * R * S * T);
  hipMalloc((void**)&d_O, sizeof(float) * K * M * P * Q * N);
  float* h_O = (float *)malloc(sizeof(float) * K * M * P * Q * N);
  std::cout << "before" << std::endl;

  hipError_t result = hipMemcpy(h_O, d_O, sizeof(float) * K * M * P * Q * N, hipMemcpyDeviceToHost);
  if (result != hipSuccess) {
    std::cout << result << std::endl;
    std::cerr << "memcpy error!" << std::endl;
  }

  for (int i = 0; i < 10; ++i) {
    std::cout << h_O[i] << " ";
  }
  std::cout << std::endl;

  hipMemcpy(d_I, h_I, sizeof(float) * N * C * D * H * W,
    hipMemcpyHostToDevice);
  hipMemcpy(d_F, h_F, sizeof(float) * K * R * S * T,
    hipMemcpyHostToDevice);

  if (!load_kernels("./")) {
    std::cerr << "Couldn't load all kernels" << std::endl;
    exit(1);
  }


  if (!fprop(d_I, d_F, d_O, N, C, K, D, H, W, R, S, T, M, P, Q, str_d, str_h, str_w, pad_d, pad_h, pad_w)) {
    std::cerr << "Launch error" << std::endl;
  }

  std::cout << "result" << std::endl;

  result = hipMemcpy(h_O, d_O, sizeof(float) * K * M * P * Q * N, hipMemcpyDeviceToHost);
  if (result != hipSuccess) {
    std::cout << result << std::endl;
    std::cerr << "memcpy error!" << std::endl;
  }

  for (int i = 0; i < 10; ++i) {
    std::cout << h_O[i] << " ";
  }
  
  std::cout << std::endl;

  free(h_O);
  free(h_I);
  free(h_F);
  hipFree(d_I);
  hipFree(d_F);
  hipFree(d_O);

  std::cout << "finish" << std::endl;

  return 0;
}
