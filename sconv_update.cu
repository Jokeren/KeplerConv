#include "sconv.h"

bool update(const float *I, float *F, const float *O,
  unsigned int N, unsigned int C, unsigned int K,
  unsigned int D, unsigned int H, unsigned int W,
  unsigned int R, unsigned int S, unsigned int T,
  unsigned int M, unsigned int P, unsigned int Q,
  unsigned int str_d, unsigned int str_h, unsigned int str_w,
  unsigned int pad_d, unsigned int pad_h, unsigned int pad_w) {
  float alpha = 1.0f;
  unsigned int DHW, WN, HW, HWN, DHWN, CRST, RST, RS;
  unsigned int PQ, QN, PQN, MPQN;
  unsigned int magic_HW, magic_W;
  unsigned int shift_HW, shift_W;
  unsigned int magic_RST, magic_RS, magic_S;
  unsigned int shift_RST, shift_RS, shift_S;
  unsigned int magic_PQu, shift_PQu;
  unsigned int magic_Qu, shift_Qu;
  unsigned int grid_P = 1;
  unsigned int grid_Q = 1;
  unsigned int grid_PQ = grid_P * grid_Q;
  unsigned int grid_PQM = grid_PQ * M;

  WN = W * N;
  HW = H * W;
  HWN = H * WN;
  DHW = D * HW;
  DHWN = D * HWN;
  RS = R * S;
  RST = T * RS;
  CRST = C * RS;

  QN = Q * N;
  PQN = P * QN;
  MPQN = M * PQN;

  magic32(CRST, RST, magic_RST, shift_RST);
  magic32(RST + 32, RS, magic_RS, shift_RS);
  magic32(RS + 32, S, magic_S, shift_S);
  magic32(DHW, HW, magic_HW, shift_HW);
  magic32(HW, W, magic_W, shift_W);
  magic32(grid_PQM, grid_PQ, magic_PQu, shift_PQu);
  magic32(grid_PQ, grid_Q, magic_Qu, shift_Qu);

  float *test_param;
  hipError_t result;
  result = hipMalloc((void**)&test_param, sizeof(float) * 1024);
  hipMemset(test_param, 0, sizeof(float) * 1024);
  if (result != hipSuccess) {
    std::cerr << "cuda malloc error!" << std::endl;
    exit(1);
  }
  void *args[43] = {&test_param, &I, &O, &F, &alpha,
    &N, &K, &D, &H, &W, &WN, &HWN, &DHWN, &C, &CRST,
    &RST, &magic_RST, &shift_RST, &RS, &magic_RS, &shift_RS, &S, &magic_S, &shift_S,
    &pad_d, &pad_h, &pad_w, &str_d, &str_h, &str_w,
    &P, &Q, &PQ, &QN, &PQN, &MPQN,
    &magic_Qu, &shift_Qu, &magic_PQu, &shift_PQu,
    &grid_P, &grid_Q, &grid_PQ};

  int gridX = grid_PQM;
  int gridY = CRST / 128 + (CRST % 128 != 0);
  int gridZ = N / 64 + (N % 64 != 0);

  std::string name = "sconv_update_C128_K128";
  hipError_t res = hipModuleLaunchKernel(nervana_kernels[name], gridX, gridY, gridZ, 256, 1, 1, R * S * T * 4 * 2, 0, args, NULL);
  if (res != hipSuccess) {
    std::cerr << "Error launching kernel " << name << " " << res << std::endl;
    return false;
  }
  hipCtxSynchronize();
  //float* h_test = (float *)malloc(sizeof(float) * 64);
  //for (int i = 0; i < 64; ++i) {
  //  std::cout << h_test[i] << " ";
  //}
  //std::cout << std::endl;
  //result = hipMemcpy(h_test, test_param, sizeof(float) * 64, hipMemcpyDeviceToHost);
  //if (result != hipSuccess) {
  //  std::cout << result << std::endl;
  //  std::cerr << "memcpy error!" << std::endl;
  //}

  //for (int i = 0; i < 64; ++i) {
  //  std::cout << h_test[i] << " ";
  //}

  //free(h_test);

  //std::cout << std::endl;

  return true;
}

int main() {
  hipFree(0);
  float *d_I, *d_F, *d_O;
  unsigned int N = 64, C = 1, K = 128, D = 1, H = 5, W = 5, T = 1, R = 5, S = 5;
  unsigned int str_d = 1, str_h = 1, str_w = 1;
  unsigned int pad_d = 0, pad_h = 0, pad_w = 0;
  unsigned int M, P, Q;
  M = (D - T + 2 * pad_d) / str_d + 1;
  P = (H - R + 2 * pad_h) / str_h + 1;
  Q = (W - S + 2 * pad_w) / str_w + 1;
  float *h_O = (float *)malloc(N * M * P * Q * K * sizeof(float));
  for (int i = 0; i < N * M * P * Q * K; ++i) {
    h_O[i] = 1;
  }
  float *h_I = (float *)malloc(K * D * H * W * N * sizeof(float));
  for (int i = 0; i < K * D * H * W * N; ++i) {
    h_I[i] = 1;
  }
  float* h_F = (float *)malloc(sizeof(float) * K * R * S * T);
  hipMalloc((void**)&d_I, sizeof(float) * N * C * D * H * W);
  hipMalloc((void**)&d_F, sizeof(float) * K * R * S * T);
  hipMalloc((void**)&d_O, sizeof(float) * K * M * P * Q * N);
  hipMemcpy(d_I, h_I, sizeof(float) * N * C * D * H * W,
    hipMemcpyHostToDevice);
  hipMemcpy(d_O, h_O, sizeof(float) * N * M * P * Q * K,
    hipMemcpyHostToDevice);

  if (!update(d_I, d_F, d_O, N, C, K, D, H, W, R, S, T, M, P, Q, str_d, str_h, str_w, pad_d, pad_h, pad_w)) {
    std::cerr << "Launch error" << std::endl;
  }

  std::cout << "result" << std::endl;

  hipError_t result = hipMemcpy(h_F, d_F, sizeof(float) * C * K * R * S * T, hipMemcpyDeviceToHost);
  if (result != hipSuccess) {
    std::cout << result << std::endl;
    std::cerr << "memcpy error!" << std::endl;
  }

  for (int i = 0; i < 10; ++i) {
    std::cout << h_F[i] << " ";
  }
  
  std::cout << std::endl;

  free(h_O);
  free(h_I);
  free(h_F);
  hipFree(d_I);
  hipFree(d_F);
  hipFree(d_O);

  std::cout << "finish" << std::endl;

  return 0;
}
