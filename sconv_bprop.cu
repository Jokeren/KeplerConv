#include "sconv.h"

bool bprop(float *I, const float *F, const float *I,
  unsigned int N, unsigned int C, unsigned int K,
  unsigned int D, unsigned int H, unsigned int W,
  unsigned int R, unsigned int S, unsigned int T,
  unsigned int M, unsigned int P, unsigned int Q,
  unsigned int str_d, unsigned int str_h, unsigned int str_w,
  unsigned int pad_d, unsigned int pad_h, unsigned int pad_w) {
  float alpha = 1.0f;
  unsigned int DHW, WN, HW, HWN, DHWN, CRST, RST, RS;
  unsigned int MPQ, PQ, QN, PQN, MPQN;
  unsigned int magic_HW, magic_W;
  unsigned int shift_HW, shift_W;
  unsigned int magic_RST, magic_RS, magic_S;
  unsigned int shift_RST, shift_RS, shift_S;
  unsigned int magic_Q, shift_Q, magic_PQ, shift_PQ;
  unsigned int magic_str_w, magic_str_h, magic_str_d;
  unsigned int shift_str_w, shift_str_h, shift_str_d;

  WN = W * N;
  HW = H * W;
  HWN = H * WN;
  DHW = D * HW;
  DHWN = D * HWN;
  RS = R * S;
  RST = T * RS;
  CRST = C * RS;

  QN = Q * N;
  PQN = P * QN;
  MPQN = M * PQN;
  PQ = P * Q;
  MPQ = M * P * Q;

  magic32(MPQ, PQ, magic_PQ, shift_PQ);
  magic32(PQ, Q, magic_Q, shift_Q);
  magic32(CRST, RST, magic_RST, shift_RST);
  magic32(RST + 32, RS, magic_RS, shift_RS);
  magic32(RS + 32, S, magic_S, shift_S);
  magic32(W + S - pad_w - 2, str_w, magic_str_w, shift_str_w);
  magic32(H + R - pad_h - 2, str_h, magic_str_h, shift_str_h);
  magic32(D + T - pad_d - 2, str_d, magic_str_d, shift_str_d);
  magic32(DHW, HW, magic_HW, shift_HW);
  magic32(HW, W, magic_W, shift_W);

  float *test_param;
  hipError_t result;
  result = hipMalloc((void**)&test_param, sizeof(float) * 1024);
  hipMemset(test_param, 0, sizeof(float) * 1024);
  if (result != hipSuccess) {
    std::cerr << "cuda malloc error!" << std::endl;
    exit(1);
  }
  void *args[40] = {&test_param, &I, &O, &F, &alpha,
    &N, &C, &M, &P, &Q, &QN, &PQN, &MPQN,
    &K, &CRST, &RST, &RS, &magic_RS, &shift_RS, &S, &magic_S, &shift_S,
    &pad_d, &pad_h, &pad_w, &str_d, &str_h, &str_w,
    &W, &HW, &WN, &HWN, &DHWN, &magic_W, &shift_HW,
    &R, &T, &magic_str_w, &magic_str_h, &magic_str_d};
  int gridDWH = D * W * H;
  int gridX = gridDWH;
  int gridY = C / 64 + (C % 64 != 0);
  int gridZ = N / 64 + (N % 64 != 0);

  std::string name = "sconv_fprop_K64_N64";
  hipError_t res = hipModuleLaunchKernel(nervana_kernels[name], gridX, gridY, gridZ, 64, 1, 1, R * S * T * 4 * 2, 0, args, NULL);
  if (res != hipSuccess) {
    std::cerr << "Error launching kernel " << name << " " << res << std::endl;
    return false;
  }
  hipCtxSynchronize();
  //float* h_test = (float *)malloc(sizeof(float) * 64);
  //for (int i = 0; i < 64; ++i) {
  //  std::cout << h_test[i] << " ";
  //}
  //std::cout << std::endl;
  //result = hipMemcpy(h_test, test_param, sizeof(float) * 64, hipMemcpyDeviceToHost);
  //if (result != hipSuccess) {
  //  std::cout << result << std::endl;
  //  std::cerr << "memcpy error!" << std::endl;
  //}

  //for (int i = 0; i < 64; ++i) {
  //  std::cout << h_test[i] << " ";
  //}

  //free(h_test);

  //std::cout << std::endl;

  return true;
}

int main() {
  hipFree(0);
  float *d_I, *d_F, *d_O;
  unsigned int N = 64, C = 64, K = 1, D = 1, H = 5, W = 5, T = 1, R = 5, S = 5;
  unsigned int str_d = 1, str_h = 1, str_w = 1;
  unsigned int pad_d = 0, pad_h = 0, pad_w = 0;
  unsigned int M, P, Q;
  M = (D - T + 2 * pad_d) / str_d + 1;
  P = (H - R + 2 * pad_h) / str_h + 1;
  Q = (W - S + 2 * pad_w) / str_w + 1;
  float *h_O = (float *)malloc(N * M * P * Q * K * sizeof(float));
  for (int i = 0; i < N * M * P * Q * K; ++i) {
    h_O[i] = 1;
  }
  float *h_F = (float *)malloc(K * R * S * T * sizeof(float));
  for (int i = 0; i < K * R * S * T; ++i) {
    h_F[i] = 1;
  }
  float* h_I = (float *)malloc(sizeof(float) * K * M * P * Q * N);
  hipMalloc((void**)&d_I, sizeof(float) * N * C * D * H * W);
  hipMalloc((void**)&d_F, sizeof(float) * K * R * S * T);
  hipMalloc((void**)&d_O, sizeof(float) * K * M * P * Q * N);
  hipMemcpy(d_O, h_O, sizeof(float) * N * M * P * Q * K,
    hipMemcpyHostToDevice);
  hipMemcpy(d_F, h_F, sizeof(float) * K * R * S * T,
    hipMemcpyHostToDevice);

  if (!bprop(d_I, d_F, d_O, N, C, K, D, H, W, R, S, T, M, P, Q, str_d, str_h, str_w, pad_d, pad_h, pad_w)) {
    std::cerr << "Launch error" << std::endl;
  }

  std::cout << "result" << std::endl;

  hipError_t result = hipMemcpy(h_I, d_I, sizeof(float) * C * D * H * W * N, hipMemcpyDeviceToHost);
  if (result != hipSuccess) {
    std::cout << result << std::endl;
    std::cerr << "memcpy error!" << std::endl;
  }

  for (int i = 0; i < 10; ++i) {
    std::cout << h_I[i] << " ";
  }
  
  std::cout << std::endl;

  free(h_O);
  free(h_I);
  free(h_F);
  hipFree(d_I);
  hipFree(d_F);
  hipFree(d_O);

  std::cout << "finish" << std::endl;

  return 0;
}
