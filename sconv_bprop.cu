#include "sconv.h"

std::string kernel_name = "sconv_bprop_C64_N64";

bool bprop(float *I, const float *F, const float *O,
  unsigned int N, unsigned int C, unsigned int K,
  unsigned int D, unsigned int H, unsigned int W,
  unsigned int R, unsigned int S, unsigned int T,
  unsigned int M, unsigned int P, unsigned int Q,
  unsigned int str_d, unsigned int str_h, unsigned int str_w,
  unsigned int pad_d, unsigned int pad_h, unsigned int pad_w) {
  float alpha = 1.0f;
  unsigned int DHW, WN, HW, HWN, DHWN, CRST, RST, RS;
  unsigned int MPQ, PQ, QN, PQN, MPQN;
  unsigned int magic_HW, magic_W;
  unsigned int shift_HW, shift_W;
  unsigned int magic_RST, magic_RS, magic_S;
  unsigned int shift_RST, shift_RS, shift_S;
  unsigned int magic_Q, shift_Q, magic_PQ, shift_PQ;
  unsigned int magic_str_w, magic_str_h, magic_str_d;
  unsigned int shift_str_w, shift_str_h, shift_str_d;
  // input
  WN = W * N;
  HW = H * W;
  HWN = H * WN;
  DHW = D * HW;
  DHWN = D * HWN;
  // filter
  RS = R * S;
  RST = T * RS;
  CRST = C * RS;
  // output
  QN = Q * N;
  PQN = P * QN;
  MPQN = M * PQN;
  PQ = P * Q;
  MPQ = M * P * Q;
  // magic numbers
  magic32(MPQ, PQ, magic_PQ, shift_PQ);
  magic32(PQ, Q, magic_Q, shift_Q);
  magic32(CRST, RST, magic_RST, shift_RST);
  magic32(RST + 32, RS, magic_RS, shift_RS);
  magic32(RS + 32, S, magic_S, shift_S);
  magic32(W + S - pad_w - 2, str_w, magic_str_w, shift_str_w);
  magic32(H + R - pad_h - 2, str_h, magic_str_h, shift_str_h);
  magic32(D + T - pad_d - 2, str_d, magic_str_d, shift_str_d);
  magic32(DHW, HW, magic_HW, shift_HW);
  magic32(HW, W, magic_W, shift_W);
  // test param set up
  float *test_param;
  hipError_t cuda_error;
  cuda_error = hipMalloc((void**)&test_param, sizeof(float) * 1024);
  hipMemset(test_param, 0, sizeof(float) * 1024);
  void *args[45] = {
    &test_param, &I, &O, &F, &alpha,
    &N, &C, &M, &P, &Q, &QN, &PQN, &MPQN,
    &K, &CRST, &RST,
    &RS, &magic_RS, &shift_RS,
    &S, &magic_S, &shift_S,
    &pad_d, &pad_h, &pad_w,
    &str_d, &str_h, &str_w,
    &W, &HW, &WN, &HWN, &DHWN,
    &magic_W, &shift_W,
    &magic_HW, &shift_HW,
    &R, &T,
    &magic_str_w, &shift_str_w,
    &magic_str_h, &shift_str_h,
    &magic_str_d, &shift_str_d};
  int gridDWH = D * W * H;
  int gridX = gridDWH;
  int gridY = C / 64 + (C % 64 != 0);
  int gridZ = N / 64 + (N % 64 != 0);
  hipError_t res = hipModuleLaunchKernel(nervana_kernels[kernel_name], gridX, gridY, gridZ, 64, 1, 1,
    RST * 4 * 2, 0, args, NULL);
  if (res != hipSuccess) {
    std::cerr << "Line " << __LINE__ << " error launching kernel " << kernel_name << " " << res << std::endl;
    return false;
  }
  hipCtxSynchronize();
  float* h_test = (float *)malloc(sizeof(float) * 64);
  for (int i = 0; i < 64; ++i) {
    std::cout << h_test[i] << " ";
  }
  std::cout << std::endl;
  cuda_error = hipMemcpy(h_test, test_param, sizeof(float) * 64, hipMemcpyDeviceToHost);
  if (cuda_error != hipSuccess) {
    std::cerr << "Line " << __LINE__ << " memcpy error: " << cuda_error << std::endl;
    exit(1);
  }
  for (int i = 0; i < 64; ++i) {
    std::cout << h_test[i] << " ";
  }
  std::cout << std::endl;
  // free test_param
  free(h_test);
  return true;
}

int main() {
  // init
  hipFree(0);
  // params
  float *d_I, *d_F, *d_O;
  unsigned int N = 64, C = 64, K = 3, D = 1, H = 5, W = 5, T = 1, R = 5, S = 5;
  unsigned int str_d = 1, str_h = 1, str_w = 1;
  unsigned int pad_d = 0, pad_h = 0, pad_w = 0;
  unsigned int M, P, Q;
  hipError_t cuda_error;
  M = (D - T + 2 * pad_d) / str_d + 1;
  P = (H - R + 2 * pad_h) / str_h + 1;
  Q = (W - S + 2 * pad_w) / str_w + 1;
  float *h_O = (float *)malloc(K * M * P * Q * N * sizeof(float));
  for (int i = 0; i < K * M * P * Q * N; ++i) {
    h_O[i] = 1;
  }
  float *h_F = (float *)malloc(K * R * S * T * C * sizeof(float));
  for (int i = 0; i < K * R * S * T * C; ++i) {
    h_F[i] = 1;
  }
  float* h_I = (float *)malloc(sizeof(float) * C * D * H * W * N);
  // device memory
  hipMalloc((void**)&d_I, sizeof(float) * C * D * H * W * N);
  hipMalloc((void**)&d_F, sizeof(float) * K * R * S * T * C);
  hipMalloc((void**)&d_O, sizeof(float) * K * M * P * Q * N);
  // memcpy h_O, h_F
  hipMemcpy(d_O, h_O, sizeof(float) * M * P * Q * K * N,
    hipMemcpyHostToDevice);
  hipMemcpy(d_F, h_F, sizeof(float) * K * R * S * T * C,
    hipMemcpyHostToDevice);
  // load kernels 
  if (!load_kernels("./")) {
    std::cerr << "Couldn't load all kernels" << std::endl;
    exit(1);
  }
  // launch kernel
  if (!bprop(d_I, d_F, d_O, N, C, K, D, H, W, R, S, T, M, P, Q, str_d, str_h, str_w, pad_d, pad_h, pad_w)) {
    std::cerr << "Launch error" << std::endl;
    exit(1);
  }
  // output
  std::cout << "result" << std::endl;
  cuda_error = hipMemcpy(h_I, d_I, sizeof(float) * C * D * H * W * N, hipMemcpyDeviceToHost);
  if (cuda_error != hipSuccess) {
    std::cerr << "Line " << __LINE__ << " memcpy error: " << cuda_error << std::endl;
    exit(1);
  }
  for (int i = 0; i < 128; ++i) {
    std::cout << h_I[i] << " ";
  }
  std::cout << std::endl;
  // free memory
  free(h_O);
  free(h_I);
  free(h_F);
  hipFree(d_I);
  hipFree(d_F);
  hipFree(d_O);
  // run successfully
  std::cout << "finish" << std::endl;
  return 0;
}
